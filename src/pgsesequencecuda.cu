#include "hip/hip_runtime.h"
#include "pgsesequencecuda.cuh"
#include "constants.h"
#include "Eigen/Dense"
#include <math.h>
#include <algorithm>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <stdlib.h>
#include <time.h>       /* time_t, struct tm, difftime, time, mktime */
#include <assert.h>


#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

using namespace std;

__global__ void update_phase_shift_cuda(int walker_batch_size, double *traj, double *grad_sequence, int num_rep, int T, double *phase_shift_cuda){

    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int stride = blockDim.x * gridDim.x;

    /*
     - Template parameter for walker_batch_size 
     - parallel unroll
    */

    for (int idx_walker=i; idx_walker < walker_batch_size; idx_walker+=stride){

        if (idx_walker<walker_batch_size){

            int idx_time_s = idx_walker*3*(T+1); 
            double *curr_traj = &(traj[idx_time_s]);
            double pos_init[3]={curr_traj[0], curr_traj[1], curr_traj[2]};


            int idx_phase_shift_s = idx_walker*num_rep;
            double *curr_phase_shift = &(phase_shift_cuda[idx_phase_shift_s]);

            double dos_pi = 2.0*M_PI;

            double xt[3];

            double val = 0.0;
            
            int grad_sequence_idx = 0;

            for (int tt=1; tt <= T; tt++){ 
                //Displacement
                xt[0] = curr_traj[tt*3] - pos_init[0];
                xt[1] = curr_traj[tt*3+1] - pos_init[1];
                xt[2] = curr_traj[tt*3+2] - pos_init[2];

                
                for(int ss=0; ss < num_rep ; ss++){
                    grad_sequence_idx = 3*(ss*T + (tt-1)); 
                    val = (giro*(grad_sequence[grad_sequence_idx]*xt[0]+grad_sequence[grad_sequence_idx+1]*xt[1]+grad_sequence[grad_sequence_idx+2]*xt[2]));                
                    val = fmod(val, dos_pi);                    
                    curr_phase_shift[ss] = fmod(curr_phase_shift[ss]+val,dos_pi);
                }
            }
        }
    }
    return;
}


__global__ void update_DWI_signal_cuda(int num_rep, double *phase_shift_cuda, int walker_batch_size, double* DWI_cuda){

    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int stride = blockDim.x * gridDim.x;

    for (int idx_phase_shift=i; idx_phase_shift<num_rep;idx_phase_shift+=stride){

        //cudaMemCheck --> Check no outer-range index

        if (idx_phase_shift<num_rep){

            for (int j=idx_phase_shift; j<(idx_phase_shift+(num_rep*walker_batch_size)); j+=num_rep){

                DWI_cuda[idx_phase_shift] += cos(phase_shift_cuda[j]);
                

            }    
        }
    }
    return; // What about last elements ? idx_phase_shift - (num_rep*walker_batch_size)
    
}
  

__global__ void cudaCreateGradSequence( double time_step, int T, int num_rep, double *scheme_vec, double *grad_sequence){

	double tcurr, tlast;
    
    int acq_index = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;
    int scheme_index=0;

    for (int i=acq_index;i<num_rep;i+=stride){

        if (i<num_rep){

            scheme_index = i*7;

            double g[3]  = { scheme_vec[scheme_index],scheme_vec[scheme_index+1], scheme_vec[scheme_index+2]};
            double G     =  scheme_vec[scheme_index+3];
            double Delta =  scheme_vec[scheme_index+4];
            double delta =  scheme_vec[scheme_index+5];
            double te    =  scheme_vec[scheme_index+6];
            double pad = (te - Delta - delta)/2.0;

            double firstBlockStart = pad;
            double firstBlockEnd = pad+delta;

            double secondBlockStart = pad+Delta;
            double secondBlockEnd = pad+Delta+delta;

            double sgn = 0.0;
            double dt = 0.0;

            int grad_idx=3*i*T;


            for(int t = 1 ; t <= T; t++){

                tlast = time_step*(double)(t-1.0);
                tcurr = time_step*(double)(t);

                // impulse condition
                if (!(tcurr >= 0.0 && tcurr<=te)){
                    sgn = 0.0;
                    dt = 0.0;
                } 

                else if ( (tcurr < pad) || (tcurr > te-pad)){
                    sgn = 0.0;
                    dt = 0.0;
                }

                else if ((tcurr >=firstBlockStart) && (tcurr < firstBlockEnd)){                
                    //between pad and first block
                    sgn = 1.0;
                    if(tlast < firstBlockStart){
                        dt = tcurr - firstBlockStart;
                    }
                    else{
                        dt=time_step;
                    }
                }

                else if ((tcurr >=firstBlockEnd) && (tcurr < secondBlockStart)){            
                    //between the 2 blocks
                    if(tlast < firstBlockEnd){
                        sgn = 1.0;
                        dt= firstBlockEnd-tlast;
                    }
                    else{
                        sgn = 0.0;
                        dt  = time_step;
                    }
                }

                else if((tcurr >= secondBlockStart)&&(tcurr <= secondBlockEnd)){
                    sgn = -1.0;
                    if (tlast < secondBlockStart){
                    // the block ended between this call and the last one
                    // so need to calculate the partial contribution
                        dt  = tcurr-secondBlockStart;
                    }
                    else{
                        dt=time_step;
                    }
                }
                else if (tcurr >= secondBlockEnd){
                    if (tlast<secondBlockEnd){
                        // the block ended between this call and the last one
                        // so need to calculate the partial contribution
                        sgn = -1.0;
                        dt = secondBlockEnd-tlast;
                    }
                    else{
                        sgn = 0.0;
                        dt=time_step;
                    }
                }
                else{
                    sgn= 0.0;
                    dt=time_step;
                }
                if (sgn!=0.0){
                    for (int j=0 ; j < 3; j++){
                        grad_sequence[grad_idx+j] = sgn*G*dt*g[j];
                    }
                }
                grad_idx+=3;
                
            }// end t
        }//end grad_index 
    }  
    return;
}

__global__ void cudaInitGrad(int T, int num_rep, double* grad_sequence){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int stride = blockDim.x * gridDim.x;

    for (int idx_grad=i; idx_grad<(3*num_rep*T);idx_grad+=stride){
        if (idx_grad<(3*num_rep*T)){
            grad_sequence[idx_grad]=0.0;
        }
    }
}

__global__ void cudaInitDWI(int walker_batch_size, int num_rep, double* DWI_cuda){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int stride = blockDim.x * gridDim.x;

    for (int idx_phase_shift=i; idx_phase_shift<num_rep;idx_phase_shift+=stride){
        DWI_cuda[idx_phase_shift]=0.0;
    }
}

__global__ void cudaInitPhaseShift(int walker_batch_size, int num_rep, double *phase_shift_cuda){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int stride = blockDim.x * gridDim.x;

    for (int idx_walker=i; idx_walker < (walker_batch_size*num_rep); idx_walker+=stride){
        phase_shift_cuda[idx_walker]=0.0;
    }    
}

PGSESequenceCuda::PGSESequenceCuda()
{
    num_rep = 0;
    dynamic = false;
    save_phase_shift = true;
    percent_steps_in = -1;
    T = 0;
    separate_signal=false;
}


PGSESequenceCuda::PGSESequenceCuda(Scheme scheme_)
{
    num_rep=0;
    dynamic = false;
    save_phase_shift = true;
    percent_steps_in = -1;
    readSchemeParameters(scheme_);
    phase_shift_distribution.resize(num_rep,3600);
    phase_shift_distribution = Eigen::ArrayXXf::Zero(num_rep,3600);
}




PGSESequenceCuda::~PGSESequenceCuda()
{
}

void PGSESequenceCuda::getGradImpulse(int grad_index, double t, double tLast, Eigen::Vector3d& Gdt){

    for(int i = 0; i < 3; i++)
        Gdt[i] = 0;

    double g[3]  = {scheme[grad_index][0],scheme[grad_index][1],scheme[grad_index][2]};
    double G     =  scheme[grad_index][3];
    double Delta =  scheme[grad_index][4];
    double delta =  scheme[grad_index][5];
    double te    =  scheme[grad_index][6];

    //printf("%.25f - %.25f - %.25f - %.25f - %.25f - %.25f - %.25f - \n",g[0],g[1],g[2],G,Delta,delta,te );
    //cout << " " << g[0] << " " << g[1] << " " << g[2] << " " << G << " " << Delta << " " << delta << " " << te << endl;

    if (!(t >= 0.0 && t<=te)){
        return;
    }

    //    printf("%d - %.25f - %.25f \n",grad_index,t,tLast);

    double pad = (te - Delta - delta)/2.0;
    if ( (t < pad) || (t > te-pad)){
        return;
    }

    double firstBlockStart = pad;
    double firstBlockEnd = pad+delta;
    //between pad and first block
    double sgn = 1;
    if( t >=firstBlockStart && t < firstBlockEnd){
        if(tLast < firstBlockStart){
            double dt = t - firstBlockStart;
            for (int j=0; j < 3; j++){
                Gdt[j] = sgn*G*dt*g[j];
            }
            return;
        }
    }

    double secondBlockStart = pad+Delta;
    double secondBlockEnd = pad+Delta+delta;

    //between the 2 blocks
    sgn = 1;
    if( t >=firstBlockEnd && t < secondBlockStart){
        if(tLast < firstBlockEnd){
            double dt= firstBlockEnd-tLast;
            for (int j=0; j < 3; j++){
                Gdt[j] = sgn*G*dt*g[j];
            }
            return;
        }
        return;
    }

    //segundo bloque
    if (t >= secondBlockStart){
        sgn=-1;
    }

    //if after second block
    if (t >= secondBlockEnd){
        if (tLast<secondBlockEnd){
            // the block ended between this call and the last one
            // so need to calculate the partial contribution
            double dt = secondBlockEnd-tLast;
            for (int j=0; j < 3; j++){
                Gdt[j] = sgn*G*dt*g[j];
            }
            return;
        }
        return;
    }

    if((t >= secondBlockStart)&&( tLast < secondBlockStart)){
        // the block ended between this call and the last one
        // so need to calculate the partial contribution
        double dt= t-secondBlockStart;

        for (int j=0; j < 3; j++){
            Gdt[j] = sgn*G*dt*g[j];
        }
        return;
    }
    for (int j=0 ; j < 3; j++){
        Gdt[j] = sgn*G*(t-tLast)*g[j];
    }
}


void PGSESequenceCuda::readSchemeParameters(Scheme scheme_){

    scheme_file = scheme_.scheme_file;
    dyn_duration = scheme_.scheme[0][6];

    num_rep = scheme_.scheme.size();

    for(unsigned i = 0 ; i < num_rep; i++){
        DWI.push_back(0);
        DWIi.push_back(0);

        phase_shift.push_back(0);
        scheme.push_back(scheme_.scheme[i]);
    }
}

void PGSESequenceCuda::readSchemeFile()
{
    ifstream in(scheme_file.c_str());

    //TODO: Error handling
    if(!in.is_open()){
        cout << "[ERROR] Can't open the scheme file " << endl;
        in.close();
        return;
    }

    vector<double> scheme_line;
    double tmp;
    string header;
    in >> header;
    in >> header;
    num_rep = 0;
    while( in >> tmp){
        scheme_line.push_back(tmp);
        DWI.push_back(0);

        if(this->img_signal == true)
            DWIi.push_back(0);

        if(separate_signal){
            DWI_extra.push_back(0);
            DWI_intra.push_back(0);
        }

        num_rep++;
        for(int i = 0 ; i < 6; i++){
            in >> tmp;
            scheme_line.push_back(tmp);
        }
        scheme.push_back(scheme_line);
        scheme_line.clear();
    }

    in.close();
}


void PGSESequenceCuda::createGradSequence(double *grad_sequence){

	double time_step	= this->dyn_duration/this->T;
	double tcurr(0.0), tlast(0.0);
    
    int grad_idx=0;

    double g[3]  = {0.0, .0, .0};
    double G     =  0;
    double Delta =  0;
    double delta =  0;
    double te    =  0;
    double pad =  0;

    double firstBlockStart =  0;
    double firstBlockEnd =  0;

    double secondBlockStart =  0;
    double secondBlockEnd =  0;

    double sgn = 0.0;
    double dt = 0.0;

    for (int grad_index=0 ; grad_index < this->num_rep ; grad_index++){

        g[0]  = this->scheme[grad_index][0];
        g[1]  = this->scheme[grad_index][1];
        g[2]  = this->scheme[grad_index][2];
        G     =  this->scheme[grad_index][3];
        Delta =  this->scheme[grad_index][4];
        delta =  this->scheme[grad_index][5];
        te    =  this->scheme[grad_index][6];
        pad = (te - Delta - delta)/2.0;

        firstBlockStart = pad;
        firstBlockEnd = pad+delta;

        secondBlockStart = pad+Delta;
        secondBlockEnd = pad+Delta+delta;

        sgn = 0.0;
        dt = 0.0;

        for(int t = 1 ; t <= T; t++){

            tlast = time_step*(double)(t-1.0);
            tcurr = time_step*(double)(t);

            // impulse condition
            if (!((tcurr >= 0.0) && (tcurr<=te))){
               sgn = 0;
               dt = 0.0;
            } 

            else if ((tcurr < pad) || (tcurr > te-pad)){
                sgn = 0;
                dt = 0.0;
            }

            else if ((tcurr >=firstBlockStart) && (tcurr < firstBlockEnd)){                
                sgn = 1;
                if(tlast < firstBlockStart){
                    //between pad and first block
                    dt = tcurr - firstBlockStart;
                }
                else{
                    dt=time_step;
                }
            }

            else if ((tcurr >=firstBlockEnd) && (tcurr < secondBlockStart)){            
                if(tlast < firstBlockEnd){
                    sgn = 1;
                    dt= firstBlockEnd-tlast;
                }
                else{
                    //between the 2 blocks
                    sgn = 0;
                    dt  = time_step;
                }
            }

            else if((tcurr >= secondBlockStart)&&(tcurr <= secondBlockEnd)){
                sgn = -1;
                if (tlast < secondBlockStart){
                // the block ended between this call and the last one
                // so need to calculate the partial contribution
                    dt  = tcurr-secondBlockStart;
                }
                else{
                    dt=time_step;
                }
            }
            else if (tcurr >= secondBlockEnd){
                if (tlast<secondBlockEnd){
                    // the block ended between this call and the last one
                    // so need to calculate the partial contribution
                    sgn = -1;
                    dt = secondBlockEnd-tlast;
                }
                else{
                    sgn = 0.0;
                    dt=0.0;
                }
            }
            else{
                sgn= 0.0;
                dt=time_step;
            }
            
            for (int j=0 ; j < 3; j++){
                grad_sequence[grad_idx+j] = (sgn*G*dt*g[j]);
            }
            grad_idx+=3;
            
        }// end t
    }//end grad_index   
}


void PGSESequenceCuda::initCudaVariables(int walker_batch_size){

    printf("Cuda init...");

    int blockSize = 256;
    int numBlocks = 0;

    hipError_t err_malloc;

    // Grad sequence 

    err_malloc = hipMallocManaged(&this->grad_sequence,  (3*(this->T)*(this->num_rep))*sizeof(double));
    
    if (err_malloc!=hipSuccess){
        printf("var %d %d ", this->T, this->num_rep);
        printf("Memory Error grad sequence");
        return;
    }
    


    /*
    numBlocks = ((3*this->num_rep*this->T) + blockSize - 1) / blockSize;

    cudaInitGrad<<<numBlocks, blockSize>>>(this->T, this->num_rep, this->grad_sequence);

    numBlocks = (this->num_rep + blockSize - 1) / blockSize;

    double time_step	= this->dyn_duration/this->T;
    int nb_elem = (this->num_rep*this->scheme[0].size());

    double scheme_vec[nb_elem];
        
    int curr_idx = 0;
    for (int curr_acq=0;curr_acq<this->num_rep;curr_acq++){
        for (int jj=0;jj<this->scheme[curr_acq].size();jj++){
            curr_idx = curr_acq*(this->scheme[curr_acq].size())+jj;
            scheme_vec[curr_idx] = (double)(this->scheme[curr_acq][jj]);
        }
    }

    double *scheme_vec_ptr = scheme_vec;

    cudaCreateGradSequence<<<numBlocks, blockSize>>>(time_step, this->T, this->num_rep, scheme_vec_ptr, this->grad_sequence);
    */

    this->createGradSequence(this->grad_sequence);

   
    // Phase shift
    err_malloc=hipMallocManaged(&this->phase_shift_cuda,  (walker_batch_size*this->num_rep)*sizeof(double));

    if (err_malloc!=hipSuccess){
        printf("Memory Error phase_shift_cuda");
        return;
    }
    numBlocks = (walker_batch_size + blockSize - 1) / blockSize;
    cudaInitPhaseShift<<<numBlocks, blockSize>>>(walker_batch_size, this->num_rep, this->phase_shift_cuda);

    //DWI signal
    err_malloc=hipMallocManaged(&this->DWI_cuda,  (this->num_rep)*sizeof(double));
    if (err_malloc!=hipSuccess){
        printf("Memory Error DWI_cuda");
        return;
    }

    numBlocks = (this->num_rep + blockSize - 1) / blockSize;
    cudaInitDWI<<<numBlocks, blockSize>>>(walker_batch_size, this->num_rep, this->DWI_cuda);
    
    // Wait synchronization
    hipDeviceSynchronize();

    printf("Done\n");

    return;

}

void PGSESequenceCuda::freeCudaVariables(){
    // Free memory
    hipFree(this->phase_shift_cuda);
    hipFree(this->grad_sequence);
    hipFree(this->DWI_cuda);
    return;
    
}

void PGSESequenceCuda::resetCudaVariables(int walker_batch_size){

    int blockSize = 1;
    int numBlocks = 1;

    cudaInitPhaseShift<<<numBlocks, blockSize>>>(walker_batch_size, this->num_rep, this->phase_shift_cuda);
    hipDeviceSynchronize();

    return;

}

void PGSESequenceCuda::update_phase_shift_DWI_signal(double* traj_mat, int walker_batch_size){   
    
    int blockSize = 256;
    int numBlocks = (walker_batch_size + blockSize - 1) / blockSize;


    printf("Phase shift update <%d, %d>...\n ", blockSize, numBlocks);
    //GPU Phase shift update
    update_phase_shift_cuda<<<numBlocks, blockSize>>>(walker_batch_size, traj_mat,  this->grad_sequence, this->num_rep, this->T, this->phase_shift_cuda);
    
    // Wait synchronization
    hipDeviceSynchronize();
    printf("Done\n");

    // GPU signal update
    numBlocks = (this->num_rep + blockSize - 1) / blockSize;

    printf("DWI update<%d, %d>...\n ", blockSize, numBlocks);
    update_DWI_signal_cuda<<<numBlocks, blockSize>>>(this->num_rep, this->phase_shift_cuda, walker_batch_size, this->DWI_cuda);

    // Wait synchronization
    hipDeviceSynchronize();
    printf("Done\n");

    for (unsigned int ii=0;ii<(this->num_rep);ii++){
        this->DWI[ii] = this->DWI_cuda[ii];
    }    
    return;
    
}



void PGSESequenceCuda::setNumberOfSteps(unsigned T)
{
    this->T = T;
}

void PGSESequenceCuda::computeDynamicTimeSteps()
{
    double Delta =  scheme[0][4];
    double delta =  scheme[0][5];
    double TE    =  scheme[0][6];
    double pad   = (TE - Delta - delta)/2.0;

    unsigned steps_in = percent_steps_in*T;

    //we want them to be even
    if(steps_in%2)
        steps_in++;

    delta = delta + delta*delta/20;

    int steps_pad = (2.0*pad)/(2.0*pad + Delta - delta) * (T-steps_in);

    //we want them to be even
    if(steps_pad%2)
        steps_pad++;

    int steps_out = T - steps_in - steps_pad;

    if( steps_in <= 0 || steps_out <= 0 || T<=0 || steps_pad <=0 || percent_steps_in <= 0){
        cout << "[Error] Incoherent number of steps inside the gradient pulse!" << endl;
        assert(0);
    }

    time_steps.resize(T+1,1);

    double dt_pad = (2*pad) / double(steps_pad);

    double dt_out = (Delta-delta)/double(steps_out);

    double dt_in  = (2.0*delta)/double(steps_in);

    ulong count    = 0.0;
    double time    = 0.0;


    for(int i=0;i < steps_pad/2.0; i++){
        time_steps[count++] = time;
        time += dt_pad;
    }


    for(int i=0;i < steps_in/2.0; i++){
        time_steps[count++] = time;
        time += dt_in;
    }


    for(int i=0;i < steps_out; i++){
        time_steps[count++] = time;
        time += dt_out;
    }

    for(int i=0;i < steps_in/2.0; i++){
        time_steps[count++] = time;
        time += dt_in;
    }

    for(int i=0;i <= steps_pad/2.0; i++){
        time_steps[count++] = time;
        time += dt_pad;
    }


    if(count != T+1){
        cout << "WARNING! T was not fullilled correctly in the dynamic setting!" <<endl;
    }

    //    for(int i = 0 ; i < T+1; i++)
    //        cout << time_steps[i] << endl;

}


double PGSESequenceCuda::getbValue(unsigned i)
{
    double G     =  scheme[i][3];
    double Delta =  scheme[i][4];
    double delta =  scheme[i][5];

    return (G*delta*giro)*(G*delta*giro)*(Delta - delta/3);
}

double PGSESequenceCuda::getFreeDecay(unsigned i,double D){
    double b = getbValue(i);

    return exp(-b*D);
}


double PGSESequenceCuda::getNumericalbValue(unsigned i)
{
    return -i;
}

void PGSESequenceCuda::getDWISignal()
{

    trajectory.initTrajReaderFile();

    trajectory.readTrajectoryHeader();

    double N        = trajectory.N;
    double T        = trajectory.T;
    double duration = trajectory.dyn_duration;
    double rt       = duration/T;
    double dos_pi   = 2.0*M_PI;
    double dt,dt_last,xt[3];

    Eigen::Matrix3Xd steps_log; // complete trajectory of one walker

    Eigen::Vector3d Gdt;
    Eigen::VectorXd phase_shift;

    steps_log.resize(3,unsigned(T+1));
    phase_shift.resize(num_rep);

    for (int w = 0; w < N; w++)
    {
        trajectory.readCurrentWalkersTrajectory(steps_log);
        for (uint t = 1; t <= uint(T); t++)
        {
            dt      = rt*(t);
            dt_last = rt*(t-1.0);

            xt[0] = steps_log(0,t) - steps_log(0,0);
            xt[1] = steps_log(1,t) - steps_log(1,0);
            xt[2] = steps_log(2,t) - steps_log(2,0);

            for(int s=0; s < num_rep ;s++)
            {
                getGradImpulse(s,dt,dt_last,Gdt);
                double val = giro*(Gdt[0]*xt[0]+Gdt[1]*xt[1]+Gdt[2]*xt[2]);

                val = fmod(val,2*M_PI);
                //printf("%d - %1.25f \n",w,val );
                phase_shift[s] = fmod(phase_shift[s]+ val,dos_pi);
            }
        }

        for(uint s=0; s < num_rep; s++){
            DWI[s] += cos(phase_shift[s]); // Real part

            if(this->img_signal == true)
                DWIi[s]+= sin(phase_shift[s]); // Img part

            phase_shift[s] = 0;
        }
    }

}// END getDWISignal

double PGSESequenceCuda::get_adt(int grad_index, double t, double tLast){

    double Delta =  scheme[grad_index][4];
    double delta =  scheme[grad_index][5];
    double te    =  scheme[grad_index][6];

    //printf("%.25f - %.25f - %.25f - %.25f - %.25f - %.25f - %.25f - \n",g[0],g[1],g[2],G,Delta,delta,te );
    //cout << " " << g[0] << " " << g[1] << " " << g[2] << " " << G << " " << Delta << " " << delta << " " << te << endl;

    if (!(t >= 0.0 && t<=te)){
        return -INFINITY_VALUE;
    }

    //    printf("%d - %.25f - %.25f \n",grad_index,t,tLast);

    double pad = (te - Delta - delta)/2.0;
    if ( (t < pad) || (t > te-pad)){
        return 0;
    }

    double firstBlockStart = pad;
    double firstBlockEnd = pad+delta;
    //between pad and first block
    double sgn = 1;
    if( t >=firstBlockStart && t < firstBlockEnd){
        if(tLast < firstBlockStart){
            double dt = t - firstBlockStart;

            return sgn*dt;
        }
    }

    double secondBlockStart = pad+Delta;
    double secondBlockEnd = pad+Delta+delta;

    //between the 2 blocks
    sgn = 1;
    if( t >=firstBlockEnd && t < secondBlockStart){
        if(tLast < firstBlockEnd){
            double dt= firstBlockEnd-tLast;
            return sgn*dt;
        }
        return 0;
    }

    //segundo bloque
    if (t >= secondBlockStart){
        sgn=-1;
    }

    //if after second block
    if (t >= secondBlockEnd){
        if (tLast<secondBlockEnd){
            // the black ended between this call and the last one
            // so need to calculate the partial contribution
            double dt = secondBlockEnd-tLast;

            return sgn*dt;
        }
        return 0 ;
    }

    if((t >= secondBlockStart)&&( tLast < secondBlockStart)){
        // the block ended between this call and the last one
        // so need to calculate the partial contribution
        double dt= t-secondBlockStart;

        return sgn*dt;;
    }

    return sgn*(t-tLast);
}
